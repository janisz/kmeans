#include "hip/hip_runtime.h"
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
# define WINDOWS_LEAN_AND_MEAN
# define NOMINMAX
# include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h> // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h> // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h> // helper functions for CUDA error check
#include <helper_cuda_gl.h> // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD 0.30f
#define REFRESH_DELAY 10 //ms

////////////////////////////////////////////////////////////////////////////////

#define MESH_SIZE (width*height)

const unsigned int window_width = 800;
const unsigned int window_height = 800;

const unsigned int width = 150;
const unsigned int height = 150;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

//Table containing velocity of all particles. It is pointer to device memory
float2 *speed = NULL;
float4 *dptr = NULL;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0; // FPS count for averaging
int fpsLimit = 1; // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runProgram(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
		unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void prepareCuda(struct hipGraphicsResource **vbo_resource);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

/*-----------------------------------------------------------------------------------------------------*/

__inline static
float euclid_dist_2(int numdims, /* no. dimensions */
                    float *coord1, /* [numdims] */
                    float *coord2) /* [numdims] */
{
    int i;
    float ans=0.0;

    for (i=0; i<numdims; i++)
        ans += (coord1[i]-coord2[i]) * (coord1[i]-coord2[i]);

    return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__inline static
int find_nearest_cluster(int numClusters, /* no. clusters */
                         int numCoords, /* no. coordinates */
                         float *object, /* [numCoords] */
                         float **clusters) /* [numClusters][numCoords] */
{
    int index, i;
    float dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    min_dist = euclid_dist_2(numCoords, object, clusters[0]);

    for (i=1; i<numClusters; i++) {
        dist = euclid_dist_2(numCoords, object, clusters[i]);
        /* no need square root */
        if (dist < min_dist) { /* find the min and its array index */
            min_dist = dist;
            index = i;
        }
    }
    return(index);
}

/*----< seq_kmeans() >-------------------------------------------------------*/
/* return an array of cluster centers of size [numClusters][numCoords] */
float** seq_kmeans(float **objects, /* in: [numObjs][numCoords] */
                   int numCoords, /* no. features */
                   int numObjs, /* no. objects */
                   int numClusters, /* no. clusters */
                   float threshold, /* % objects change membership */
                   int *membership, /* out: [numObjs] */
                   int *loop_iterations)
{
    int i, j, index, loop=0;
    int *newClusterSize; /* [numClusters]: no. objects assigned in each
new cluster */
    float delta; /* % of objects change their clusters */
    float **clusters; /* out: [numClusters][numCoords] */
    float **newClusters; /* [numClusters][numCoords] */

    //printf("/* allocate a 2D space for returning variable clusters[] (coordinates of cluster centers) */\n");
    clusters = (float**) malloc(numClusters * sizeof(float*));
    assert(clusters != NULL);
    clusters[0] = (float*) malloc(numClusters * numCoords * sizeof(float));
    assert(clusters[0] != NULL);
    for (i=1; i<numClusters; i++)
        clusters[i] = clusters[i-1] + numCoords;

    //printf("/* pick first numClusters elements of objects[] as initial cluster centers*/\n");
    for (i=0; i<numClusters; i++)
        for (j=0; j<numCoords; j++)
            clusters[i][j] = objects[i][j];

    //printf("/* initialize membership[] */\n");
    for (i=0; i<numObjs; i++) membership[i] = -1;

	//printf("/* need to initialize newClusterSize and newClusters[0] to all 0 */\n");
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    newClusters = (float**) malloc(numClusters * sizeof(float*));
    assert(newClusters != NULL);
    newClusters[0] = (float*) calloc(numClusters * numCoords, sizeof(float));
    assert(newClusters[0] != NULL);
    for (i=1; i<numClusters; i++)
        newClusters[i] = newClusters[i-1] + numCoords;

    do {
        delta = 0.0;
        for (i=0; i<numObjs; i++) {

			//printf("/* find the array index of nearest cluster center */\n");
            index = find_nearest_cluster(numClusters, numCoords, objects[i],
                                         clusters);

			//printf("/* if membership changes, increase delta by 1 */\n");
            if (membership[i] != index) delta += 1.0;

			//printf("/* assign the membership to object i */\n");
            membership[i] = index;

			//printf("/* update new cluster centers : sum of objects located within */\n");
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[index][j] += objects[i][j];
        }

        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    clusters[i][j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0; /* set back to 0 */
            }
            newClusterSize[i] = 0; /* set back to 0 */
        }

        delta /= numObjs;
    } while (delta > threshold && loop++ < 500);

    *loop_iterations = loop + 1;

    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}

/*-----------------------------------------------------------------------------------------------------*/

float randFloat(float LO, float HI)
{
	return LO + (float)rand()/((float)RAND_MAX/(HI-LO));
}

void prepare_kernel(float4 *pos, float time)
{
	for (int index = 0;index<MESH_SIZE;index++)	{
		// calculate uv coordinates
		float u = (index / width) / (float) width;
		float v = (index % width) / (float) height;
	    u = u*2-1;
	    v = v*2-1;
	    // calculate simple sine wave pattern
	    float freq = 4.0f;

	    float w = sinf(u*freq+time) * sinf(v*freq+time);

		// write output vertex
		pos[index] = make_float4(u, v, w, 0);

	}
}

int main(int argc, char **argv)
{
	char *ref_file = NULL;

	pArgc = &argc;
	pArgv = argv;

	dptr = new float4[MESH_SIZE];

	printf("starting...\n");

	runProgram(argc, argv, ref_file);

	printf("completed, returned %s\n", (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit) {
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "CPU: %3.1f FPS \t(X:%d\t Y:%d)", avgFPS, mouse_old_x, mouse_old_y);
	glutSetWindowTitle(fps);
}

bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent,0);

	// initialize necessary OpenGL extensions
	glewInit();

	if (! glewIsSupported("GL_VERSION_2_0 ")) {
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	return true;
}

void prepareCuda()
{
	prepare_kernel(dptr, 0);
}

int *membership = NULL;
float** obj = NULL;

void runKmeans()
{
	//launch_kernel(dptr, width, height, speed);
	if (membership == NULL) {
		membership = new int[height*width];
	}

	static double t;
	t += 0.01;
	prepare_kernel(dptr, t);

	if (obj == NULL) {
		obj = new float*[height*width];
	}

	for (int i=0;i<height*width;i++) {
		obj[i] = (float*)(&dptr[i]);
	}
	int loops;

	seq_kmeans(obj, /* in: [numObjs][numCoords] */
			   4, 	/* no. features */
			   width * height, /* no. objects */
			   3, 	/* no. clusters */
			   0.01, /* % objects change membership */
			   membership, /* out: [numObjs] */
			   &loops);
}

bool runProgram(int argc, char **argv, char *ref_file)
{
	prepareCuda();
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv)) {
		return false;
	}

	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);


	// start rendering mainloop
	glutMainLoop();
	atexit(cleanup);

	return true;
}

void setGLColorForCluster(int index)
{
	if (membership[index] == 0)
		glColor3f( 1, 0, 0 );
	if (membership[index] == 1)
		glColor3f( 0, 1, 0 );
	if (membership[index] == 2)
		glColor3f( 0, 0.5, 1 );
	if (membership[index] == 3)
			glColor3f( 0, 1, 1 );
	if (membership[index] == 4)
			glColor3f( 1, 1, 0 );
}

void display()
{
	sdkStartTimer(&timer);

	runKmeans();
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 0.0, 1.0);

	glColor3f( 1, 1, 1 );
	glutWireCube (2.0);


	glBegin( GL_POINTS );
	for ( int i = 0; i < MESH_SIZE; ++i )
	{
		setGLColorForCluster(i);
		glPointSize(5);
		glVertex3f( dptr[i].x, dptr[i].y, dptr[i].z );
	}
	glEnd();
	glFinish();
	glutSwapBuffers();

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	glutPostRedisplay();
	glutTimerFunc(REFRESH_DELAY, timerEvent,0);
}

void cleanup()
{
	sdkDeleteTimer(&timer);
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            exit(EXIT_SUCCESS);
            break;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}
