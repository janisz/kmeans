#include "hip/hip_runtime.h"
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
# define WINDOWS_LEAN_AND_MEAN
# define NOMINMAX
# include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h> // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h> // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h> // helper functions for CUDA error check
#include <helper_cuda_gl.h> // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD 0.30f
#define REFRESH_DELAY 10 //ms

#define DBG			(fprintf(stderr,"%s:%d:\t",__FILE__,__LINE__));

////////////////////////////////////////////////////////////////////////////////
// constants
#define MAX_DISTANCE 0.005
#define MIN_DISTANCE 0.001
#define MAX_ANGLE M_PI
#define MAX_SPEED 0.09
#define W_NEIGHBOUR_SPEED 0.15
#define W_NEIGHBOUR_DISTANCE 0.15
#define W_MIN_DISTANCE 0.15
#define W_NOISE 0.1
#define COMPARE(x, y) (((x) > (y)) - ((x) < (y)))
#define SIGN(x) COMPARE(x, 0)
const unsigned int window_width = 800;
const unsigned int window_height = 800;

const unsigned int mesh_width = 20;
const unsigned int mesh_height = 20;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

//Table containing velocity of all particles. It is pointer to device memory
float2 *speed = NULL;
float4 *dptr = NULL;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = -3.0, rotate_y = 3.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0; // FPS count for averaging
int fpsLimit = 1; // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
		unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void prepareCuda(struct hipGraphicsResource **vbo_resource);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

/*-----------------------------------------------------------------------------------------------------*/

__inline static
float euclid_dist_2(int numdims, /* no. dimensions */
                    float *coord1, /* [numdims] */
                    float *coord2) /* [numdims] */
{
    int i;
    float ans=0.0;

    for (i=0; i<numdims; i++)
        ans += (coord1[i]-coord2[i]) * (coord1[i]-coord2[i]);

    return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__inline static
int find_nearest_cluster(int numClusters, /* no. clusters */
                         int numCoords, /* no. coordinates */
                         float *object, /* [numCoords] */
                         float **clusters) /* [numClusters][numCoords] */
{
    int index, i;
    float dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    min_dist = euclid_dist_2(numCoords, object, clusters[0]);

    for (i=1; i<numClusters; i++) {
        dist = euclid_dist_2(numCoords, object, clusters[i]);
        /* no need square root */
        if (dist < min_dist) { /* find the min and its array index */
            min_dist = dist;
            index = i;
        }
    }
    return(index);
}

/*----< seq_kmeans() >-------------------------------------------------------*/
/* return an array of cluster centers of size [numClusters][numCoords] */
float** seq_kmeans(float **objects, /* in: [numObjs][numCoords] */
                   int numCoords, /* no. features */
                   int numObjs, /* no. objects */
                   int numClusters, /* no. clusters */
                   float threshold, /* % objects change membership */
                   int *membership, /* out: [numObjs] */
                   int *loop_iterations)
{
    int i, j, index, loop=0;
    int *newClusterSize; /* [numClusters]: no. objects assigned in each
new cluster */
    float delta; /* % of objects change their clusters */
    float **clusters; /* out: [numClusters][numCoords] */
    float **newClusters; /* [numClusters][numCoords] */

    printf("/* allocate a 2D space for returning variable clusters[] (coordinates of cluster centers) */\n");
    clusters = (float**) malloc(numClusters * sizeof(float*));
    assert(clusters != NULL);
    clusters[0] = (float*) malloc(numClusters * numCoords * sizeof(float));
    assert(clusters[0] != NULL);
    for (i=1; i<numClusters; i++)
        clusters[i] = clusters[i-1] + numCoords;

    printf("/* pick first numClusters elements of objects[] as initial cluster centers*/\n");
    for (i=0; i<numClusters; i++)
        for (j=0; j<numCoords; j++)
            clusters[i][j] = objects[i][j];

    printf("/* initialize membership[] */\n");
    for (i=0; i<numObjs; i++) membership[i] = -1;

	printf("/* need to initialize newClusterSize and newClusters[0] to all 0 */\n");
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    newClusters = (float**) malloc(numClusters * sizeof(float*));
    assert(newClusters != NULL);
    newClusters[0] = (float*) calloc(numClusters * numCoords, sizeof(float));
    assert(newClusters[0] != NULL);
    for (i=1; i<numClusters; i++)
        newClusters[i] = newClusters[i-1] + numCoords;

    do {
        delta = 0.0;
        for (i=0; i<numObjs; i++) {

			printf("/* find the array index of nearest cluster center */\n");
            index = find_nearest_cluster(numClusters, numCoords, objects[i],
                                         clusters);

            DBG
			printf("/* if membership changes, increase delta by 1 */\n");
            if (membership[i] != index) delta += 1.0;

            DBG
			printf("/* assign the membership to object i */\n");
            membership[i] = index;

            DBG
			printf("/* update new cluster centers : sum of objects located within */\n");
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[index][j] += objects[i][j];
        }

        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    clusters[i][j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0; /* set back to 0 */
            }
            newClusterSize[i] = 0; /* set back to 0 */
        }

        delta /= numObjs;
    } while (delta > threshold && loop++ < 500);

    *loop_iterations = loop + 1;

    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}

/*-----------------------------------------------------------------------------------------------------*/


inline __host__ __device__ float4 operator+(const float4 &a, const float4 &b)
{
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}
inline __host__ __device__ float2 operator+(const float2 &a, const float2 &b)
{
	return make_float2(a.x + b.x, a.y + b.y);
}
inline __host__ __device__ float4 operator-(const float4 &a, const float4 &b)
{
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w);
}
inline __host__ __device__ float dot(float4 a, float4 b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w;
}
inline __host__ __device__ float distance(float4 pt1, float4 pt2)
{
	float4 v = pt2 - pt1;
	return sqrt(dot(v,v));
}
inline __host__ __device__ float FastArcTan(float x)
{
	return M_PI_4*x - x*(fabs(x) - 1)*(0.2447 + 0.0663*fabs(x));
}
inline __device__ __host__ float random(float seed)
{
	int x = 88675123;
	int y = 362436069;
	int z = 521288629;

	x = (y *((int) seed) + z) % x;

	return sinf(x);
}
struct equalOperator {
	__host__ __device__
	bool operator()(const float4 x, const float4 y) const {
		return ( x.x > y.x );
	}
};

void calculate_conditions(float4 position, float2 velocity,
		float4 *pos, float2 *speed, float2 &neighboursAvgSpeed,
		float4 &neighboursAvgPosition, int &neighboursCount,
		float &neighboursAvgDistance, int i, int index)
{
	float4 p = pos[i];
	float d = distance(position, p);

	if (d > MAX_DISTANCE) return;

	float k1 = 0, k2 = 0;

	if (velocity.x != 0)
		k1 = FastArcTan(velocity.y/velocity.x);
	if (p.x-position.x)
		k2 = FastArcTan((p.y-position.y)/(p.x-position.x));

	if (abs(k1-k2) > MAX_ANGLE) return;

	float2 s = speed[i];
	neighboursCount++;
	neighboursAvgSpeed = neighboursAvgSpeed + s;
	neighboursAvgPosition = neighboursAvgPosition + p;
	neighboursAvgDistance += d;

	if (d < MIN_DISTANCE) {
		velocity.x -= W_MIN_DISTANCE*(((p.x-position.x)*MIN_DISTANCE/d)-(p.x - position.x));//neighboursCount;
		velocity.y -= W_MIN_DISTANCE*(((p.y-position.y)*MIN_DISTANCE/d)-(p.y - position.y));///neighboursCount;
	}
}

void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float4 mousePos, float2 *speed)
{
	int N = width * height;
	for (int index = 0;index<N;index++)	{
		float4 position = pos[index];
		float2 velocity = speed[index];

		int neighboursCount = 1;
		float2 neighboursAvgSpeed = make_float2(0, 0);
		float4 neighboursAvgPosition = make_float4(0, 0, 0, 0);
		float neighboursAvgDistance = 0;

		for (int i=0; i<N && distance(position, pos[i]) <= MAX_DISTANCE; i++) {
			if (i == index) continue;
			calculate_conditions(position, velocity,
					pos, speed, neighboursAvgSpeed,
					neighboursAvgPosition, neighboursCount,
					neighboursAvgDistance, i, index);
		}

		neighboursAvgSpeed.x = neighboursAvgSpeed.x / neighboursCount;
		neighboursAvgSpeed.y = neighboursAvgSpeed.y / neighboursCount;

		neighboursAvgPosition.x = neighboursAvgPosition.x / neighboursCount;
		neighboursAvgPosition.y = neighboursAvgPosition.y / neighboursCount;

		neighboursAvgDistance /= neighboursCount;

		velocity.x += W_NEIGHBOUR_SPEED*(neighboursAvgSpeed.x - velocity.x);
		velocity.y += W_NEIGHBOUR_SPEED*(neighboursAvgSpeed.y - velocity.y);

		velocity.x += W_NEIGHBOUR_DISTANCE*(neighboursAvgPosition.x - position.x)/100.0;
		velocity.y += W_NEIGHBOUR_DISTANCE*(neighboursAvgPosition.y - position.y)/100.0;

		velocity.x += W_NOISE * (random(index/velocity.x))*MAX_SPEED;
		velocity.y += W_NOISE * (random(index/velocity.y))*MAX_SPEED;

		if (abs(velocity.x) > MAX_SPEED) velocity.x *= 0.1;
		if (abs(velocity.y) > MAX_SPEED) velocity.y *= 0.1;

		if (distance(position, mousePos) < 100*MIN_DISTANCE) velocity.x = MAX_SPEED * SIGN(position.x-mousePos.x);
		if (distance(position, mousePos) < 100*MIN_DISTANCE) velocity.y = MAX_SPEED * SIGN(position.y-mousePos.y);

		position.x += 0.1*velocity.x;
		position.y += 0.1*velocity.y;

		if (position.x > 4) {
			position.x = -4;
		}
		if (position.y > 4) {
			position.y = -4;
		}
		if (position.x < -4) {
			position.x = 4;
		}
		if (position.y < -4) {
			position.y = 4;
		}

		pos[index] = position;
		speed[index] = velocity;
	}
}

void prepare_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float2 *speed)
{
	for (int index = 0;index<width*height;index++)	{
		// calculate uv coordinates
		float u = (index / width) / (float) width;
		float v = (index % width) / (float) height;
		u = u*4.0f - 2.0f;
		v = v*4.0f - 2.0f;

		// write output vertex
		pos[index] = make_float4(u, v, 4*rand()/(float)RAND_MAX - 2, 1.0f);

	}
}

void prepare_kernel(float4 *pos, unsigned int mesh_width,
		unsigned int mesh_height, float2 *speed)
{
	prepare_vbo_kernel(pos, mesh_width, mesh_height, speed);
}

int main(int argc, char **argv)
{
	char *ref_file = NULL;

	pArgc = &argc;
	pArgv = argv;

	speed = new float2[mesh_width*mesh_height];
	dptr = new float4[mesh_width*mesh_height];

	printf("starting...\n");

	runTest(argc, argv, ref_file);

	hipDeviceReset();
	printf("completed, returned %s\n", (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit) {
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "CPU: %3.1f FPS \t(X:%d\t Y:%d)", avgFPS, mouse_old_x, mouse_old_y);
	glutSetWindowTitle(fps);
}

bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent,0);

	// initialize necessary OpenGL extensions
	glewInit();

	if (! glewIsSupported("GL_VERSION_2_0 ")) {
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	return true;
}

void prepareCuda()
{
	prepare_kernel(dptr, mesh_width, mesh_height, speed);
}

int *membership = NULL;

float** obj = NULL;

void runCuda()
{
	//launch_kernel(dptr, mesh_width, mesh_height, speed);
	if (membership == NULL) {
		membership = new int[mesh_height*mesh_width];
	}

	if (obj == NULL) {
		obj = new float*[mesh_height*mesh_width];
		for (int i=0;i<mesh_height*mesh_width;i++) {
			//obj[i] = new float[4];
				obj[i] = (float*)(&dptr[i]);
		}
	}

	int loops;


	seq_kmeans(obj, /* in: [numObjs][numCoords] */
	                   4, /* no. features */
	                   mesh_width * mesh_height, /* no. objects */
	                   4, /* no. clusters */
	                   0.1, /* % objects change membership */
	                   membership, /* out: [numObjs] */
	                   &loops);
	printf("Loops: %d", loops);
}

bool runTest(int argc, char **argv, char *ref_file)
{
	prepareCuda();
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv)) {
		return false;
	}


	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);


	// start rendering mainloop
	glutMainLoop();
	atexit(cleanup);

	return true;
}

void setGLColor(int index)
{
	if (membership[index] == 0)
		glColor3f( 1, 0, 0 );
	if (membership[index] == 1)
		glColor3f( 0, 1, 0 );
	if (membership[index] == 2)
		glColor3f( 0, 0, 1 );
	if (membership[index] == 3)
			glColor3f( 0, 1, 1 );
	if (membership[index] == 4)
			glColor3f( 1, 1, 0 );
}

void display()
{
	sdkStartTimer(&timer);

	runCuda();
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	glBegin( GL_POINTS );
	glPointSize(15);
	for ( int i = 0; i < mesh_width*mesh_height; ++i )
	{
		setGLColor(i);
		glVertex3f( dptr[i].x, dptr[i].y, dptr[i].z );
	}
	glEnd();
	glFinish();
	glutSwapBuffers();

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	glutPostRedisplay();
	glutTimerFunc(REFRESH_DELAY, timerEvent,0);
}

void cleanup()
{
	sdkDeleteTimer(&timer);
}


void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key) {
	case (27) :
		exit(EXIT_SUCCESS);
	break;
	}
}

void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN) {
		mouse_buttons |= 1<<button;
	} else if (state == GLUT_UP) {
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dy = y - mouse_old_y;
	float dx = x - mouse_old_y;
	if (mouse_buttons & GLUT_LEFT_BUTTON) {
		translate_z += dy * 0.01f;
	} if (mouse_buttons & GLUT_MIDDLE_BUTTON) {
		rotate_x += dx * 0.01f;
	}  if (mouse_buttons & GLUT_RIGHT_BUTTON) {
		rotate_y += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}
